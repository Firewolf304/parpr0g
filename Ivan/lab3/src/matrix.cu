#include "hip/hip_runtime.h"
#include "../include/includes.h"
class matrix_gpu;
__device__ static void getFromThread(int degree, int * counter, int n, int stringSize, int size) {
    int index_x = threadIdx.x + blockIdx.x * blockDim.x;
    int index_y = threadIdx.y + blockIdx.y * blockDim.y;
    int index_z = threadIdx.z + blockIdx.z * blockDim.z;
    int value = index_x + index_y * blockDim.y + index_z * blockDim.z;
    if(value > size ) return;
    int output = 0;
    int schet = 1;
    int count = 0;
    while(value) {
        output += schet * (value % degree);
        value /= degree;
        schet *= 10;
        count++;
    }
    if(output > 9) {
        int previousDigit = output % 10;
        output /= 10;
        while (output > 0) {
            int currentDigit = output % 10;
            output /= 10;
            if (currentDigit > previousDigit) {
                return;
            }
            previousDigit = currentDigit;
        }
    }
    atomicAdd(counter, 1);
}
__global__ void kernel(int str_size, int sub_size, int * count, int sizer) {
    getFromThread(str_size - sub_size + 1, count, sub_size, str_size, sizer);
}

class matrix_gpu {
public:
    matrix_gpu(int n, string str = "xyxxz") {
        this->str = str;
        this->n = n;
    }
    string str;
    int n = 3;
    thrust::device_vector<string> map_gpu;
    void get_number() {
        if(this->str.length() < this->n) {return;}
        if(this->n == 0) {cout << "Result GPU: "<< this->n << endl; return;}
        int * pointer;
        unsigned long long count = 0;
        hipMalloc(&pointer, sizeof(count));
        hipMemcpy(pointer, &count, sizeof(count), hipMemcpyHostToDevice);
        int size = pow(this->str.length() - this->n + 1, this->n);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, 0);
        int threads_per_block = std::pow(2, str.length() - this->n );
        int blocks_per_grid = (size + threads_per_block - 1) / threads_per_block;
        if (blocks_per_grid > deviceProp.maxThreadsPerBlock) {
            blocks_per_grid = deviceProp.maxThreadsPerBlock;
            threads_per_block = (size + blocks_per_grid - 1) / blocks_per_grid;
        }
        kernel  <<<dim3(blocks_per_grid, 1, 1), dim3( threads_per_block, 1, 1 )>>> (this->str.length(), this->n, pointer, size);
        hipDeviceSynchronize();
        hipMemcpy(&count, pointer, sizeof(count), hipMemcpyDeviceToHost);
        hipFree(pointer);
        if(count == 0) { // sorry, but for large numbers
            count = tgamma( (double)(this->str.length() + 1) ) / (tgamma( (double)(n + 1) ) * tgamma(  (double)(this->str.length() - n + 1) ) );
        }
        cout << "Result GPU: "<< count << endl;
    }
};
class matrix_cpu {
public:
    matrix_cpu(int n, string str = "xyxxz") {
        this->n = n;
        this->str=str;
    }
    std::vector<string> map;
    int n = 3;
    string str;
    void get_number() {
        bool skep = false;
        if(this->str.length() == this->n) { cout << "Result CPU: "<< 1 << endl; skep = true;}
        if(this->str.length() < this->n) {skep = true;}

        int count = 0;
        if(this->str.length() == this->n) {}
        std::vector<int> offset (this->n,0);
        std::vector<int> iter (this->n,0);
        int maxSIZE = this->str.length() - this->n;
        string line;
        for(int i = 0; i < this->n; i++) { line.push_back(' '); }
        for(int i = 0; i < this->n; i++) {offset[i] = i; line[i] = str[i]; }
        int c = 0;
        this->map.push_back("");
        for(int i = 0; i < this->n; i++ ) {
            this->map.back() += this->str[offset[i] + iter[i]];
        }
        count++;
        for(;c != this->n && !skep;) {
            if(iter.back() + 1 <= maxSIZE) {
                iter.back()++;
            }
            else {
                for (int i = 0; i < n; i++) {
                    if (iter[i] >= maxSIZE) {
                        if(iter[i - 1] + 1 <= maxSIZE) {
                            iter[i - 1]++;
                        }
                        if (iter[i - 1] <= maxSIZE) {
                            iter[i] = iter[i - 1];
                        }
                        for (int j = i; j < n; j++) {
                            iter[j] = iter[i];
                        }
                    }
                }
            }
            this->map.push_back("");
            for(int i = 0; i < this->n; i++ )
                this->map.back() += this->str[offset[i] + iter[i]];
            c = 0;
            for(auto d : iter) {
                if(d == maxSIZE) {
                    c++;
                }
            }
            count++;
        }
        cout << "Result CPU: "<< count << endl;
    }
};
