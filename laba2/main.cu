#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <functional>
#include <vector>
#include <cmath>
#include <string>
#include <fstream>
#include <omp.h>
//#include "matrixer.h"
using std::cout;
using std::cin;
using std::endl;
using std::vector;
class make_cube;
__global__ void MakeMat_byKernel(make_cube * obj , thrust::device_ptr<float> data);
__global__ void collect_byKernel(make_cube * obj , thrust::device_ptr<float> data, float* output, int x, int y, int z);
struct dataId {
    struct th {
        unsigned int x;
        unsigned int y;
        unsigned int z;
    } thread;
    struct bl {
        unsigned int x;
        unsigned int y;
        unsigned int z;
    } block;
    struct blDim {
        unsigned int x;
        unsigned int y;
        unsigned int z;
    } blockDim;
};
/*__global__ void test(thrust::device_ptr<int> H) {

    int a = 1 + 1;
    (H[0][0]) = a;
}
int main() {

    //std::cout << "Hello, World!" << std::endl;
    int N = 5;
    thrust::device_vector<int> H(N);

    test<<<1, 1>>>( thrust::device_pointer_cast(H.data()) );
    cout << H[0] << endl;
    return 0;
}
*/



/*
Нынешнее представление:
                    +--------+                   представление координат
                   /|       /|                    +------------------+
                  / |      / |                    |  y               |
Координата y -> y+--------+  |                    |  ^    z          |
                 |  |     |  |                    |  |   ^           |
  Координата z ->| z+-----|--+                    |  |  /            |
                 | /      | /                     |  | /             |
                 |/       |/                      |  |/              |
                 0--------+x    <- координата x   |  0----------> x  |
              (0;0;0)                             +------------------+
              радиус цилиндра считается по стороне x z от координат центра
              T1 находится на стороне YZ при Z=0
              T2 находится на обратной стороне YZ при Z=len(Z)-1
     */

    class cpu {
        typedef vector<vector<vector<double>>> matrix_type;
    public:

        cpu(int x, int y, int z){
            this->x = x;
            this->y = y;
            this->z = z;
        }
        cpu(int x, int y, int z, double cylinder_radius) : cpu(x,y,z) {
            /*if(cylinder_radius > (float)(x/2) && cylinder_radius > (float)(y/2) && cylinder_radius > (float)(z/2))  {
                throw std::runtime_error("Cylinder is out of cube");
            }*/
            this->radius = cylinder_radius;
            matrix = matrix_type(x, vector<vector<double>>(y,vector<double>(z, 0)));
        }
        ~cpu() {
            this->matrix.clear();
        }
        std::string file_name = "cpumatrix";
        int x=2,y=2,z=2, radius = 2;
        double T1 = 20.0f;              // температура 1 стороны
        double T2 = 10.0f;              // температура противоположной стороны
        double T_bottom = 0.0f;         // температура нижней грани
        double alpha = 0.05351f;           // коэф теплопроводности
        bool show_iter = false;
        /*-----matrix-----*/
        matrix_type matrix;
        void computeMat() {
            for(int i = 0; i < this->x; i++) {
                for(int j = 0; j < this->y; j++) {
                    for(int k = 0; k < this->z; k++) {
                        if(k == 0) {
                            this->matrix[i][j][k] = this->T1;
                        }
                        else if(k == this->z-1) {
                            this->matrix[i][j][k] = this->T2;
                        }
                        /*else {
                            if(std::sqrt( std::pow( i - (this->x / 2) , 2 ) + std::pow( j - (this->y / 2) , 2 ) + std::pow( k - (this->z / 2) , 2 ) ) )
                        }*/
                    }
                }
            }
        }
        void genValue() {
            srand(time(NULL));
            this->T1 = 100 - (rand() % 50);
            this->T2 = 100 - (rand() % 50);
        }
        void iteration (double eps = 0.0001f, int maxIteration=10000) { // using basic matrix iterator

            double norm = 1;
            int count = 0;
            while (norm > eps && count < maxIteration) {
                // перебор каждой точки
                for(int i = 0; i < this->x; i++) {
                    for (int j = 0; j < this->y; j++) {
                        for (int k = 0; k < this->z; k++) {

                            // просмотр соседей и сумма новой температуры
                            double oldTemp = this->matrix[i][j][k];
                            double newTemp = 0.0f;
                            if(this->show_iter){ cout << "newtemp = "; }
                            for(int x1 = ((i>0) ? i-1 : i) ; x1 <= i+1 && x1 < this->x; x1++) {
                                for(int y1 = ((j>0) ? j-1 : j); y1 <= j+1 && y1 < this->y; y1++) {
                                    for(int z1 = ((k>0) ? k-1 : k); z1 <= k+1 && z1 < this->z; z1++) {

                                        if (this->radius < (float) std::sqrt(std::pow(x1 - (float) (this->x / 2), 2) +
                                                                             std::pow(z1 - (float) (this->z / 2),
                                                                                      2)) /*&& std::abs(y1 - (this->y / 2)) <= (this->y / 2)*/) {
                                            if(this->show_iter){cout << " + " << this->matrix[x1][y1][z1];}
                                            newTemp += this->matrix[x1][y1][z1];
                                        }

                                    }
                                }
                            }
                            //newTemp *= this->alpha; // формула распределения: dT/dt = alpha * ( (d^2T/dx^2) + (d^2T/dy^2) + (d^2T/dz^2) )
                            // т.е. нужно взять от зависимой точки сумму всех соседей и умножить на коэф распределения

                            if(this->show_iter) {cout << endl;}
                            this->matrix[i][j][k] = newTemp * this->alpha;
                            if(this->show_iter) {cout << "newtemp=" << newTemp << " matrix=" << this->matrix[i][j][k] << " (" << i << "," << j << "," << k << ")" << endl;}
                            norm = std::abs( newTemp - oldTemp );
                        }
                    }
                }
                count++;
            }

        }
        void show() {
            for (int i = 0; i < this->x; i++) {
                for (int k = 0; k < z; k++) {
                    cout << this->matrix [i][y-1][k] << " ";
                }
                cout << endl;
            }
            cout << "-------------" << endl;
            for (int j = 0; j < y; j++) {
                for (int i = 0; i < x; i++) {
                    cout << this->matrix[i][j][(int)(z/2)] << " ";
                }
                cout << endl;
            }
            cout << "-------------" << endl;
            for (int j = 0; j < y; j++) {
                for (int k = 0; k < z; k++) {
                    cout << this->matrix[(int)(x/2)][j][k] << " ";
                }
                cout << endl;
            }
            cout << "-------------" << endl;
            for (int j = 0; j < y; j++) {
                for (int i = 0; i < x; i++) {
                    cout << this->matrix[i][j][(int)(z-1)] << " ";
                }
                cout << endl;
            }
            cout << "-------------" << endl;
            for (int j = 0; j < y; j++) {
                for (int i = 0; i < x; i++) {
                    cout << this->matrix[i][j][(int)(0)] << " ";
                }
                cout << endl;
            }
        }

        void visualize() {
            /*
             Пример отрисовки:
             Матрицы 3 на 3:
             1 0 1
             0 0 0
             1 0 1 все подобны

             отрисовка результат:

                 |   +---------+
                 |  / 1  0  1 /|
                 | / 0  0  0 /1|
                 |/ 1  0  1 /00|
               y +---------+101+
                 | 1  0  1 |00/
                 | 0  0  0 |1/
                 | 1  0  1 |/
        ---------+---------+x----------
                /|
               / |
              z  |




             метод итерации по оси xz :
             for (int i = 0, x_pos = z-1; i < x; i++, x_pos--) { // z-1 т.к. мы рассматриваем высоту строки по X в зависимости от координаты Z, т.е. кол-во строк равно Z, как и их высота
                for (int k = 0; k < z; k++) {
                    if(x_pos>=0 || i == 0 || i = x-1) {      // добавить пробелы, пока есть строки по xz или это начало строки
                        cout << " ";
                    }
                    else{
                        cout << answer[i][y-1][k] << " "; // y-1 т.к. смотрим только вверх
                    }
                }
                cout << endl;
            }
             */

        }
        matrix_type operator() () { // output matrix
            return this->matrix;
        }
    private:

    };

     __global__ class make_cube {
    private:
        __host__ __device__ int index(int x, int y, int z) {
            return x + this->x * (y + this->y * z);
            //return x * (this->y * this->z) + y * this->x + z;
        }

    public:
         std::string file_name = "gpumatrix";
        thrust::device_vector<float> matrix;
        int x=2,y=2,z=2, radius = 2;
        float T1 = 20.0f;              // температура 1 стороны
        float T2 = 10.0f;              // температура противоположной стороны
        float T_bottom = 0.0f;         // температура нижней грани
        float alpha = 0.05351f;           // коэф теплопроводности
        float eps = 0.01f;
        int maxIteration=50;
        bool run_test = false;                  // Ручной откладчик потоков и блоков
        bool show_iter = false;                 // Ручной откладчик итерации
        __host__ void genValue() {
            srand(time(NULL));
            this->T1 = 100 - (rand() % 50);
            this->T2 = 100 - (rand() % 50);
        }
        __host__ thrust::device_vector<float> operator() () { // output matrix
            return this->matrix;
        }
        __host__ __device__ make_cube() { }
        __host__ __device__ make_cube(int x, int y, int z){
            this->x = x;
            this->y = y;
            this->z = z;
        }
        __host__ __device__ ~make_cube() {
            this->matrix.shrink_to_fit();
            //this->matrix.clear();

        }

        __host__ void machine_info() {
            int deviceCount;
            hipGetDeviceCount(&deviceCount);
            cout << "Detected " << deviceCount << " devices:" << endl;
            for (int device = 0; device < deviceCount; ++device) {
                hipDeviceProp_t deviceProp;
                hipGetDeviceProperties(&deviceProp, device);
                cout << "Device " << device << ": " << deviceProp.name << endl;
                cout << "  Compute Capability: " << deviceProp.major << "." << deviceProp.minor << endl;
                cout << "  Total Global Memory: " << deviceProp.totalGlobalMem / (1024 * 1024) << " MB" << endl;
                cout << "  Multiprocessors: " << deviceProp.multiProcessorCount << endl;
                cout << "  Max Threads per Block: " << deviceProp.maxThreadsPerBlock << endl;
                cout << "  Max Threads per Multiprocessor: " << deviceProp.maxThreadsPerMultiProcessor << endl;
                cout << "  Warp Size: " << deviceProp.warpSize << endl;
                cout << "  Max Blocks per Grid: " << deviceProp.maxGridSize[0] << " x " << deviceProp.maxGridSize[1] << " x " << deviceProp.maxGridSize[2] << endl;
                cout << "  Max Threads per Dim: " << deviceProp.maxThreadsDim[0] << " x " << deviceProp.maxThreadsDim[1] << " x " << deviceProp.maxThreadsDim[2] << endl;
                cout << "  Memory Clock Rate: " << deviceProp.memoryClockRate / 1e3 << " MHz" << endl;
                cout << "  Memory Bus Width: " << deviceProp.memoryBusWidth << " bits" << endl;
                cout << "  L2 Cache Size: " << deviceProp.l2CacheSize / 1024 << " KB" << endl;
                cout << "  Max Shared Memory per Block: " << deviceProp.sharedMemPerBlock / 1024 << " KB" << endl;
                cout << "  Max Registers per Block: " << deviceProp.regsPerBlock << endl;
                cout << "  Clock Rate: " << deviceProp.clockRate / 1e3 << " MHz" << endl;
                cout << "  Texture Alignment: " << deviceProp.textureAlignment << " bytes" << endl;
                cout << "  GPU Overlap: " << (deviceProp.deviceOverlap ? "Yes" : "No") << endl;
                cout << "  Kernel Execution Timeout: " << (deviceProp.kernelExecTimeoutEnabled ? "Yes" : "No") << endl;
                cout << "  Concurrent Kernels: " << (deviceProp.concurrentKernels ? "Yes" : "No") << endl;
                cout << "  ECC Memory: " << (deviceProp.ECCEnabled ? "Yes" : "No") << std::endl;
                cout << "  Unified Addressing: " << (deviceProp.unifiedAddressing ? "Yes" : "No") << endl;
            }

        }

        __host__ make_cube(int x, int y, int z, float cylinder_radius) : make_cube(x, y, z) {
            if(cylinder_radius > (float)(x/2) || cylinder_radius > (float)(z/2))  {
                throw std::runtime_error("Cylinder is out of cube");
            }
            this->radius = cylinder_radius;
            matrix = thrust::device_vector<float>(x*y*z, 0);
            //matrix.resize(x*y*z);
            //std::fill(matrix.begin(), matrix.end(),0);
            //float* pd_vec = thrust::raw_pointer_cast(matrix.data());
            //pd_vec[0] = 20;
        }
        /*
         Паралельное заполнение элементов, логика такова:
            - 1 поток в строке(x) и столбце(y), а 256 на каждую долготу(z)
            - один блок в столбце(y) и строке(x), несколько блоков в глубине(z)
        */
        void computeMat() {
            thrust::device_ptr<float> ptr = thrust::device_pointer_cast<float>(this->matrix.data());
            make_cube * obj;
            hipMalloc(&obj, sizeof(make_cube));
            hipMemcpy(obj, this, sizeof(make_cube), hipMemcpyHostToDevice);

            //MakeMat_byKernel<<< dim3(16,16,16), dim3((this->x+ 15) / 16, (this->y+ 15) / 16, (this->z+ 15) / 16) >>>(obj, ptr );
            dim3 blocks(2, 2, 2);
            MakeMat_byKernel<<<
            dim3(
                    (this->x+ blocks.x-1) / blocks.x,
                    (this->y+ blocks.y-1) / blocks.y,
                    (this->z+ blocks.z-1) / blocks.z),
            blocks
            >>>(obj, ptr );

            if(this->run_test) {
                std::cerr << "----------------BLOCK TEST----------------" << std::endl;
                hipDeviceSynchronize();
                sleep(1);
                std::cerr << "VERIFY: BLOCKS=" << ((this->x+ blocks.x-1) / blocks.x) * ((this->y+ blocks.y-1) / blocks.y) * ((this->z+ blocks.z-1) / blocks.z)
                          << " THREADS=" << blocks.x*blocks.y*blocks.z << endl;
                std::cerr << "----------------END BLOCK TEST----------------" << std::endl;
            }
            hipMemcpy(this, obj, sizeof(make_cube), hipMemcpyDeviceToHost);
            hipFree(obj);
            hipDeviceSynchronize();

        }
        void iterateMat() {
            //int N = this->y*this->x*this->z;
            auto collector = [this](int i, int j, int k ) -> float {  // вызов kernel из host
                float value = 0;
                float * value_ptr;
                hipMalloc(&value_ptr, sizeof(float));
                hipMemcpy(value_ptr, &value, sizeof(float), hipMemcpyHostToDevice);

                thrust::device_ptr<float> ptr = thrust::device_pointer_cast<float>(this->matrix.data());
                make_cube * obj;
                hipMalloc(&obj, sizeof(make_cube));
                hipMemcpy(obj, this, sizeof(make_cube), hipMemcpyHostToDevice);
                /*
                  for(int x1 = ((i>0) ? i-1 : i) ; x1 <= i+1 && x1 < this->x; x1++) {
                                for(int y1 = ((j>0) ? j-1 : j); y1 <= j+1 && y1 < this->y; y1++) {
                                    for(int z1 = ((k>0) ? k-1 : k); z1 <= k+1 && z1 < this->z; z1++) {
                 */
                //MakeMat_byKernel<<< dim3(this->x, this->y-1, this->z), dim3((this->x+ 15) / 16, (this->y+ 15) / 16, (this->z+ 15) / 16) >>>(obj, ptr );
                dim3 blocks(1,1,1);

                collect_byKernel <<<
                dim3(
                        ((i > 0) ? 3 : 2), // Исправлено на 3, чтобы учесть случай, когда i равно 0
                        ((j > 0) ? 3 : 2), // Исправлено на 3, чтобы учесть случай, когда j равно 0
                        ((k > 0) ? 3 : 2)  // Исправлено на 3, чтобы учесть случай, когда k равно 0
                ),blocks

                >>>(obj, ptr, value_ptr, ((i>0)? i-1 : i), ((j>0)? j-1 : j), ((k>0)? k-1 : k) );
                /*dim3 blocks(2,2,3);
                collect_byKernel <<<blocks,
                dim3(
                        1,
                        1,
                        1
                )
                >>>(obj, ptr, value_ptr, ((i>0)? i-1 : i), ((j>0)? j-1 : j), ((k>0)? k-1 : k) );*/
                hipDeviceSynchronize();
                hipMemcpy(this, obj, sizeof(make_cube), hipMemcpyDeviceToHost);
                hipMemcpy(&value, value_ptr, sizeof(float), hipMemcpyDeviceToHost);                   // еблан забывать float?
                hipFree(obj);
                hipFree(value_ptr);

                return value;
            };
            if(this->run_test) {
                hipDeviceSynchronize();
                std::cerr << "----------------BLOCK TEST----------------" << std::endl;
                hipDeviceSynchronize();
                //sleep(1);
                int i=0,j=3,k=0;    // test input
                int test = collector(i, j, k);
                hipDeviceSynchronize();
                sleep(1.2);
                std::cerr << "VERIFY BLOCKS: " << (i>0 ? 3 : 2) * (j>0 ? 3 : 2) * (k>0 ? 3 : 2) << endl;
                std::cerr << "OUT: " << test << endl;
                std::cerr << "----------------END BLOCK TEST----------------" << std::endl;
                return;
            }
            float norm = 1;
            int count = 0;
            while (norm > eps && count < maxIteration) {
                // перебор каждой точки
                for(int i = 0; i < this->x; i++) {
                    for (int j = 0; j < this->y; j++) {
                        for (int k = 0; k < this->z; k++) {
                            // просмотр соседей и сумма новой температуры
                            float oldTemp = this->matrix[index(i, j, k)];
                            if(this->show_iter) { cout << "newtemp="; }
                            float newTemp = collector(i, j, k);
                            if(this->show_iter) { cout << endl; }
                            //newTemp *= this->alpha; // формула распределения: dT/dt = alpha * ( (d^2T/dx^2) + (d^2T/dy^2) + (d^2T/dz^2) )
                            // т.е. нужно взять от зависимой точки сумму всех соседей и умножить на коэф распределения
                            this->matrix[index(i, j, k)] = newTemp * this->alpha;
                            if(this->show_iter) {
                                cout << "newtemp=" << newTemp << " matrix=" << matrix[index(i, j, k)] << " (" << i << "," << j << "," << k << ")" << endl;
                            }
                            norm = std::abs( newTemp - oldTemp );
                            //printf("----------\n");
                        }
                    }
                }
                count++;
            }

        }
        __device__ void MakeMat(thrust::device_ptr<float> mat, make_cube * obj, dataId config ) {
            int x1 = config.thread.x + config.block.x * config.blockDim.x;
            int y1 = config.thread.y + config.block.y * config.blockDim.y;
            int z1 = config.thread.z + config.block.z * config.blockDim.z;
            //printf("Getted: (%d,%d,%d)\n", x1, y1, z1);
            if(z1 == 0  ) {
                /*if(obj->run_test) {
                    printf("Getted: (%d,%d,%d)\n", x1, y1, z1);
                }*/
                mat[ index(x1, y1, z1) ] = this-> T1;
                //this->matrix[ index_device(i,j,k) ] = this->T1;
                //this->matrix[i][j][k] = this->T1;
            }
            else if(z1 == (this->z)-1) {
                /*if(obj->run_test) {
                    printf("Getted: (%d,%d,%d)\n", x1, y1, z1);
                }*/
                mat[ index(x1, y1, z1) ] = this->T2;
                //this->matrix[i][j][k] = this->T2;
            }
        }
        __device__ void collectNeighbors (thrust::device_ptr<float> mat, dataId config, float* output, int xpos, int ypos, int zpos, make_cube * obj) { // using basic matrix iterator

            int x1 = config.thread.x + config.block.x * config.blockDim.x; // допустим, коорда подана: (3,0,0), решает как 3+0, 3+1, 3+2. Если коорда (0,0,0), выдаст 0+0, 0+1
            int y1 = config.thread.y + config.block.y * config.blockDim.y;
            int z1 = config.thread.z + config.block.z * config.blockDim.z;
            if(x1+xpos >= this->x || y1+ypos >= this->y || z1+zpos >= this->z) return ;
            if(obj->run_test) {
                printf("Getted: (%d,%d,%d) + (%d,%d,%d) => (%d,%d,%d) output=%.4f checked=%d\n", x1, y1, z1, xpos, ypos, zpos, x1+xpos, y1+ypos, z1+zpos, *output,
                       this->radius < (float)std::sqrt(std::pow(x1 - (float)(this->x / 2), 2) + std::pow(z1 - (float)(this->z / 2), 2)));
            }
            if(obj->radius < (float)std::sqrt(std::pow(xpos+ x1 - (float)(obj->x / 2), 2) + std::pow(zpos + z1 - (float)(obj->z / 2), 2)))
            {
                atomicAdd(output, mat[index(xpos+x1, ypos+y1, zpos+z1)]);      // на каждом блоке отдельные адреса, нельзя просто суммировать адреса
                if(obj->show_iter) {
                    float temp = mat[index(xpos + x1, ypos + y1, zpos + z1)];
                    printf("+%.1f", temp);
                }
                //(*output) += mat[index(x1, y1, z1)];

            }
            //printf("Getted: (%d,%d,%d), out=%d\n",x1,y1,z1, output);

        }
        __host__ void show(int precision = 3 ) {
            cout.setf(std::ios::fixed);
            cout.precision(precision);
            hipDeviceSynchronize();
            for (int i = 0; i < this->x; i++) {
                for (int k = 0; k < this->z; k++) {
                    cout << this->matrix[index(i, (this->y)-1, k)] << " ";
                }
                cout << endl;
            }
            cout << "-------------" << endl;
            for (int j = 0; j < this->y; j++) {
                for (int i = 0; i < this->x; i++) {
                    cout << this->matrix[index(i, j, (int)((this->z)/2))] << " ";
                }
                cout << endl;
            }
            cout << "-------------" << endl;
            for (int j = 0; j < this->y; j++) {
                for (int k = 0; k < this->z; k++) {
                    cout << this->matrix[index((int)((this->x)/2), j, k)] << " ";
                }
                cout << endl;
            }
            cout << "-------------" << endl;
            for (int j = 0; j < this->y; j++) {
                for (int i = 0; i < this->x; i++) {
                    cout << this->matrix[index(i, j, (int)(this->z-1))] << " ";
                }
                cout << endl;
            }
            cout << "-------------" << endl;
            for (int j = 0; j < this->y; j++) {
                for (int i = 0; i < this->x; i++) {
                    cout << this->matrix[index(i, j, (int)(0))] << " ";
                }
                cout << endl;
            }
        }
        __host__ void write_file(std::string prefix = "file_", int precision = 3 ) {
            std::ofstream answer( prefix + this->file_name , std::ios_base::trunc);
            answer.setf(std::ios::fixed);
            answer.precision(precision);
            hipDeviceSynchronize();
            answer << "------Сторона XZ сверху-------" << endl;
            for (int i = 0; i < this->x; i++) {
                for (int k = 0; k < this->z; k++) {
                    answer << this->matrix[index(i, (this->y)-1, k)] << " ";
                }
                answer << endl;
            }
            answer << "-------Сторона XY в разрезе------" << endl;
            for (int j = 0; j < this->y; j++) {
                for (int i = 0; i < this->x; i++) {
                    answer << this->matrix[index(i, j, (int)((this->z)/2))] << " ";
                }
                answer << endl;
            }
            answer << "-------Сторона YZ при в разрезе------" << endl;
            for (int j = 0; j < this->y; j++) {
                for (int k = 0; k < this->z; k++) {
                    answer << this->matrix[index((int)((this->x)/2), j, k)] << " ";
                }
                answer << endl;
            }
            answer << "-------Сторона YZ при Z=0------" << endl;
            for (int j = 0; j < this->y; j++) {
                for (int i = 0; i < this->x; i++) {
                    answer << this->matrix[index(i, j, (int)(this->z-1))] << " ";
                }
                answer << endl;
            }
            answer << "-------Сторона YZ при Z=Z-1------" << endl;
            for (int j = 0; j < this->y; j++) {
                for (int i = 0; i < this->x; i++) {
                    answer << this->matrix[index(i, j, (int)(0))] << " ";
                }
                answer << endl;
            }
            answer.close();
        }
    };



__global__ void MakeMat_byKernel(make_cube * obj , thrust::device_ptr<float> data) {
    //printf("Config: blocks (%d,%d,%d), threads (%d,%d,%d), blockDim(%d,%d,%d)\n",blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y,threadIdx.z,blockDim.x,blockDim.y,blockDim.z);
    dataId config = { {threadIdx.x, threadIdx.y, threadIdx.z}, {blockIdx.x, blockIdx.y, blockIdx.z}, {blockDim.x, blockDim.y, blockDim.z} };
    //printf("Verify: blocks (%d,%d,%d), threads (%d,%d,%d), blockDim(%d,%d,%d)\n", config.block.x, config.block.y, config.block.z, config.thread.x, config.thread.y,config.thread.z,config.blockDim.x,config.blockDim.y,config.blockDim.z);
    obj->MakeMat( data, obj, config );
    if(obj->run_test) {
        int threadId = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * threadIdx.z);
        int blockId = blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z);
        int numThreads = blockDim.x * blockDim.y * blockDim.z;
        int numBlocks = gridDim.x * gridDim.y * gridDim.z;
        if (threadId == 0 && blockId == 0) {
            // Вывод информации только из одного потока в блоке 0
            printf("\nMakeMat_byKernel:\n");
            printf("\tNumber of Threads: %d\n", numThreads);
            printf("\tNumber of Blocks: %d\n", numBlocks);
            printf("\tblockDim: (%d, %d, %d)\n", blockDim.x, blockDim.y, blockDim.z);
            printf("\tgridDim: (%d, %d, %d)\n", gridDim.x, gridDim.y, gridDim.z);
            printf("\tDebug values:\n");
            printf("\t\tX=%d, Y=%d, Z=%d, radius=%d\n", obj->x, obj->y, obj->z, obj->radius);
            printf("\t\trun_test=%d, maxIteration=%d, eps=%f \n", obj->run_test, obj->maxIteration, obj->eps);
            printf("\t\talpha=%f, T1=%f, T2=%f \n", obj->alpha, obj->T1, obj->T2);
        }
    }
}
__global__ void collect_byKernel(make_cube * obj , thrust::device_ptr<float> data, float* output, int x, int y, int z ) {
    //printf("Config: blocks (%d,%d,%d), threads (%d,%d,%d)\n",blockDim.x, blockDim.y, blockDim.z, threadIdx.x, threadIdx.y,threadIdx.z);
    dataId config = { {threadIdx.x, threadIdx.y, threadIdx.z}, {blockIdx.x, blockIdx.y, blockIdx.z}, {blockDim.x, blockDim.y, blockDim.z} };
    obj->collectNeighbors( data, config, output, x,y,z, obj );
    if(obj->run_test) {
        int threadId = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * threadIdx.z);
        int blockId = blockIdx.x + gridDim.x * (blockIdx.y + gridDim.y * blockIdx.z);
        int numThreads = blockDim.x * blockDim.y * blockDim.z;
        int numBlocks = gridDim.x * gridDim.y * gridDim.z;
        if (threadId == 0 && blockId == 0) {
            printf("\ncollect_byKernel:\n");
            // Вывод информации только из одного потока в блоке 0
            printf("\tNumber of Threads: %d\n", numThreads);
            printf("\tNumber of Blocks: %d\n", numBlocks);
            printf("\tblockDim: (%d, %d, %d)\n", blockDim.x, blockDim.y, blockDim.z);
            printf("\tgridDim: (%d, %d, %d)\n", gridDim.x, gridDim.y, gridDim.z);
            printf("\tDebug values:\n");
            printf("\t\tX=%d, Y=%d, Z=%d, radius=%d\n", obj->x, obj->y, obj->z, obj->radius);
            printf("\t\trun_test=%d, maxIteration=%d, eps=%f \n", obj->run_test, obj->maxIteration, obj->eps);
            printf("\t\talpha=%f, T1=%f, T2=%f \n", obj->alpha, obj->T1, obj->T2);
        }
    }
    __syncthreads();

}



/*__device__ void oo() {

}
__global__ void kernel() {
    oo();
}*/



int main() {
    cout.setf(std::ios::fixed); cout.precision(3);
    std::cout << "Hello, World!" << std::endl;
    int x = 5, y = 3, z = 5;
    float radius = 2.0f;
    float alpha = 0.001f;
    float eps = 0.01f;
    int maxIter = 50;
    bool run_test = false;
    bool show_iter = false;
    if(x%2 != 0 || y%2 != 0 || z%2 != 0) {
        std::cerr << "WARNING: scary input (multiples of 2 are required)! " << endl;
    }

    //gpu_main<<<1,1>>>( thrust::device_pointer_cast<float>( matrix.data() ), x,y,z );
    //show();
    //cube_gpu.show();
    //kernel<<<1,1>>>();
    // =================cpu zone
    cpu cube_cpu(x, y, z, radius);
    cube_cpu.alpha = alpha;
    cube_cpu.computeMat();
    cube_cpu.iteration(eps, maxIter);
    // =================cpu zone
    // =================make_cube zone
    make_cube cube_gpu(x, y, z, radius);
    //cube_gpu.machine_info();
    cube_gpu.run_test = run_test;
    cube_gpu.show_iter = show_iter;
    cube_gpu.file_name = "gpumatrix";
    if(cube_gpu.run_test) {
        std::cerr << "******************************" << std::endl;
        std::cerr << "*       RUNNED RUN_TEST      *" << std::endl;
        std::cerr << "*  (p.s. iterator disabled)  *" << std::endl;
        std::cerr << "******************************" << std::endl;
    }
    cube_gpu.alpha = alpha; //0.05351f;
    cube_gpu.eps = eps;
    cube_gpu.maxIteration = maxIter;
    //make_cube * obj;
    cube_gpu.computeMat();
    cube_gpu.iterateMat();
    //cube_gpu.write_file("output_");
    cube_gpu.show();
    // =================make_cube zone
    return 0;
}

/*__device__ __host__ int x=5,y=5,z=5, radius = 2;
__device__ __host__ float T1 = 20.0f;              // температура 1 стороны
__device__ __host__ float T2 = 10.0f;              // температура противоположной стороны
__device__ __host__ float T_bottom = 0.0f;         // температура нижней грани
__device__ __host__ float alpha = 0.001;           // коэф теплопроводности
thrust::device_vector<float> matrix(x*y*z, 0);
__host__ void genValue() {
    srand(time(NULL));
    T1 = 100 - (rand() % 50);
    T2 = 100 - (rand() % 50);
}
__host__ __device__ int index(int PointX, int PointY, int PointZ) {
    return PointX + x * (PointY + y * z);
}
__device__ void MakeMat( thrust::device_ptr<float> mat, int x_dev, int y_dev, int z_dev) {
    for(int i = 0; i < x_dev; i++) {
        for(int j = 0; j < y_dev; j++) {
            for(int k = 0; k < z_dev; k++) {
                if(k == 0) {
                    mat[ index(i, j, k) ] = T1;
                    //this->matrix[ index_device(i,j,k) ] = this->T1;
                    //this->matrix[i][j][k] = this->T1;
                }
                else if(k == z_dev-1) {
                    mat[ index(i, j, k) ] = T2;
                    //this->matrix[i][j][k] = this->T2;
                }

            }
        }
    }
}
__host__ void show() {
    hipDeviceSynchronize();
    for (int i = 0; i < x; i++) {
        for (int k = 0; k < z; k++) {
            cout << matrix[index(i, (y)-1, k)] << " ";
        }
        cout << endl;
    }
    cout << "-------------" << endl;
    for (int j = 0; j < y; j++) {
        for (int i = 0; i < x; i++) {
            cout << matrix[index(i, j, (int)((z)/2))] << " ";
        }
        cout << endl;
    }
    cout << "-------------" << endl;
    for (int j = 0; j < y; j++) {
        for (int k = 0; k < z; k++) {
            cout << matrix[index((int)((x)/2), j, k)] << " ";
        }
        cout << endl;
    }
}

__global__ void gpu_main( thrust::device_ptr<float> data, int i, int j, int k ) {

    MakeMat(data, x,y,z);
    //show();

}*/