#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <stack>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <cmath>
/*
        Задание 17
 Найти количество всех различных последовательностей символов заданного размера в
 заданной строке с учетом порядка следования символов (например, строка xyxxz содержит
 подстроки:
        x (3), y (1), z (1) размера 1,
        xy (1), xx (3), xz (3), yx (2), yz (1) размера 2,
        xyx (2) , xyz (1), xxx (1), xxz (3), yxx (1), yxz(2) размера 3,
        xyxx (1), xyxz (2), xxxz (1), yxxz (1) размера 4,
        xyxxz (1) размера 5;
 в скобках указано количество подстрок; других подстрок нет; таким образом, подстрок размера 1 ровно 5, размера 2 – 10, размера 3 – 10 и т.д.)
 */

using std::string;
using std::cout;
using std::cin;
using std::endl;
;
//string str = "abcde";

int count = 0;
std::vector<string> map_cpu;
thrust::device_vector<string> map_gpu;
/*int cycle_cpu(string piece, int a){
    string save = piece;
    for (int i = a + 1; i < str.length(); i++) {
        piece += str[i];
        if (piece.length() >= n ){
            cout<<piece<<endl;
            count +=1;
        }
        else cycle_cpu(piece, i);
        piece = save;
    }
}*/
void cycle_cpu(string str = "xyxxz", int n = 3) {
    std::vector<int> offset (n,0);
    std::vector<int> iter (n,0);
    int maxSIZE = str.length() - n;
    string line(n,' ');
    for(int i = 0; i < n; i++) {offset[i] = i; line[i] = str[i]; }
    int c = 0;
    map_cpu.push_back("");
    for(int i = 0; i < n; i++ ) {
        map_cpu.back() += str[offset[i] + iter[i]];
    }
    count++;
    while(c != n) {
        if(iter.back() + 1 <= maxSIZE) {
            iter.back()++;
        }
        else { // остальной участок просто разбить на потоки и длины так, чтобы каждый элемент двигался от 0 д maxSIZE
            for (int i = 0; i < n; i++) {
                if (iter[i] >= maxSIZE) {
                    if(iter[i - 1] + 1 <= maxSIZE) {
                        iter[i - 1]++;
                    }
                    if (iter[i - 1] <= maxSIZE) {
                        iter[i] = iter[i - 1];
                    }
                    for (int j = i; j < n; j++) {
                        iter[j] = iter[i];
                    }
                }
            }
        }
        map_cpu.push_back("");
        for(int i = 0; i < n; i++ ) {
            map_cpu.back() += str[offset[i] + iter[i]];
        }
        c = 0;
        for(auto d : iter) {
            if(d == maxSIZE) {
                c++;
            }
        }
        count++;
    }
}

__device__ void get_values(int stepen, int n, int str_size) {

    /*int value = 2;
    int schet = 1;
    thrust::device_vector<int> & H;
    H.push_back()
    while(value) {
        output += schet * (value % stepen);
        value /= stepen;
        schet *= 10;
    }*/
}
__global__ void kernel(thrust::device_ptr<int> count, int str_size, int sub_size) {
    //get_values()
    printf("HELLO\n");
}
void cycle_gpu(string str = "xyxxz", int n = 3) {
    int count = 0;
    thrust::device_ptr<int> ptr = thrust::device_pointer_cast<int>(&count);
    hipMemcpy(ptr.get(), &count, sizeof(count), hipMemcpyHostToDevice);
    kernel  <<<
        dim3(1,1,1),
        dim3(pow(n - str.length(), n),1, 1 )
    >>>( ptr , str.length(), n);
    //hipMemcpy(ptr, thrust::raw_pointer_cast(map_gpu.data()), map_gpu.size() * sizeof(string), hipMemcpyHostToDevice);
    hipMemcpy(&count, ptr.get(), sizeof(count), hipMemcpyDeviceToHost);
    hipFree(ptr.get());
    hipDeviceSynchronize();
}


int main() {
    string piece = "";
    //cycle_cpu("",-1);
    cycle_gpu();
    for(auto d : map_cpu) {
        cout << d << endl;
    }
    /*for(int i = 0; i <= str.length() - n; i ++){
        piece = str[i];
        if (piece.length() >= n && int(piece[piece.length() - 1]) != 0){
            cout<<piece<<endl;
            count +=1;
        }
        else cycle(piece, i);
    }*/
    cout<<count;
}
