#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include <stack>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
//#include <cmath>
#include <math.h>
#include <unistd.h>

#define NS_PER_SECOND 100000000000000
/*
        Задание 17
 Найти количество всех различных последовательностей символов заданного размера в
 заданной строке с учетом порядка следования символов (например, строка xyxxz содержит
 подстроки:
        x (3), y (1), z (1) размера 1,
        xy (1), xx (3), xz (3), yx (2), yz (1) размера 2,
        xyx (2) , xyz (1), xxx (1), xxz (3), yxx (1), yxz(2) размера 3,
        xyxx (1), xyxz (2), xxxz (1), yxxz (1) размера 4,
        xyxxz (1) размера 5;
 в скобках указано количество подстрок; других подстрок нет; таким образом, подстрок размера 1 ровно 5, размера 2 – 10, размера 3 – 10 и т.д.)
 */

using std::string;
using std::cout;
using std::cin;
using std::endl;
void sub_timespec(struct timespec t1, struct timespec t2, struct timespec *td)
{
    td->tv_nsec = t2.tv_nsec - t1.tv_nsec;
    td->tv_sec  = t2.tv_sec - t1.tv_sec;
    if (td->tv_sec > 0 && td->tv_nsec < 0)
    {
        td->tv_nsec += NS_PER_SECOND;
        td->tv_sec--;
    }
    else if (td->tv_sec < 0 && td->tv_nsec > 0)
    {
        td->tv_nsec -= NS_PER_SECOND;
        td->tv_sec++;
    }
}
//string str = "abcde";

std::vector<string> map_cpu;
thrust::device_vector<string> map_gpu;

/*int cycle_cpu(string piece, int a){
    string save = piece;
    for (int i = a + 1; i < str.length(); i++) {
        piece += str[i];
        if (piece.length() >= n ){
            cout<<piece<<endl;
            count +=1;
        }
        else cycle_cpu(piece, i);
        piece = save;
    }
}*/
void cycle_cpu(string str = "xyxxz", int n = 3) {
    bool skep = false;
    if(str.length() == n) { cout << "CPU: "<< 1 << endl; skep = true;}
    if(str.length() < n) {skep = true;}

    int count = 0;
    if(str.length() == n) {}
    std::vector<int> offset (n,0);
    std::vector<int> iter (n,0);
    int maxSIZE = str.length() - n;
    string line;
    for(int i = 0; i < n; i++) { line.push_back(' '); }
    for(int i = 0; i < n; i++) {offset[i] = i; line[i] = str[i]; }
    int c = 0;
    map_cpu.push_back("");
    for(int i = 0; i < n; i++ ) {
        map_cpu.back() += str[offset[i] + iter[i]];
    }
    count++;
    for(;c != n && !skep;) {
        if(iter.back() + 1 <= maxSIZE) {
            iter.back()++;
        }
        else { // остальной участок просто разбить на потоки и длины так, чтобы каждый элемент двигался от 0 д maxSIZE
            for (int i = 0; i < n; i++) {
                if (iter[i] >= maxSIZE) {
                    if(iter[i - 1] + 1 <= maxSIZE) {
                        iter[i - 1]++;
                    }
                    if (iter[i - 1] <= maxSIZE) {
                        iter[i] = iter[i - 1];
                    }
                    for (int j = i; j < n; j++) {
                        iter[j] = iter[i];
                    }
                }
            }
        }
        map_cpu.push_back("");
        for(int i = 0; i < n; i++ ) {
            map_cpu.back() += str[offset[i] + iter[i]];
        }
        c = 0;
        for(auto d : iter) {
            if(d == maxSIZE) {
                c++;
            }
        }
        count++;
    }
    /*for(auto d : map_cpu) {
        cout << d << endl;
    }*/
    cout << "CPU: "<< count << endl;
}
__device__ class debugger {
public:
    __device__ void debug(int value, const char* message) {
        printf("%d:%d:%d => %d => %s\n", threadIdx.x,threadIdx.y, threadIdx.z, value, message);
    }
    __device__ void debug(int value) {
        printf("%d:%d:%d => %d\n", threadIdx.x,threadIdx.y, threadIdx.z, value);
    }
    __device__ void debug(const char* message) {
        printf("%d:%d:%d => %s\n", threadIdx.x,threadIdx.y, threadIdx.z, message);
    }


    __device__ debugger operator<<(const char* message) {
        debug(message);
    }
    __device__ debugger operator<<(const int value) {
        debug(value);
    }
};
__device__ debugger debug;
__device__ __host__ int factorial(int n)
{
    return (n==1 || n==0) ? 1: n * factorial(n - 1);
}

__device__ void get_values(int stepen, int * counter, int n, int str_size, int sizer) {
    //int value = threadIdx.x + blockIdx.x * blockDim.x; // very small
    int index_x = threadIdx.x + blockIdx.x * blockDim.x;
    int index_y = threadIdx.y + blockIdx.y * blockDim.y;
    int index_z = threadIdx.z + blockIdx.z * blockDim.z;
    //int value = index_x + index_y * gridDim.x * blockDim.x + index_z * gridDim.x * blockDim.x * blockDim.y * gridDim.y;
    //int value = index_x + index_y * blockDim.x + index_z * blockDim.x * blockDim.y;
    int value = index_x + index_y * blockDim.y + index_z * blockDim.z;
    int save = value;
    if(value > sizer ) return;
    int output = 0;
    int schet = 1;
    int count = 0;

    while(value) {
        output += schet * (value % stepen);
        value /= stepen;
        schet *= 10;
        count++;
    }
    int copy = output;
    if(output > 9) {
        int previousDigit = output % 10;
        output /= 10;
        while (output > 0) {
            int currentDigit = output % 10;
            output /= 10;
            if (currentDigit > previousDigit) {
                return;
            }
            previousDigit = currentDigit;
        }
    }
    //printf("X:%d + Y:%d * Z:%d = %d | stepen=%d out=%d value=%d %d\n", threadIdx.x,threadIdx.y, threadIdx.z, value, stepen, copy, save, sizer );
    atomicAdd(counter, 1);
}
__global__ void kernel(int str_size, int sub_size, int * count, int sizer) { //
    //get_values()
    get_values(str_size - sub_size + 1, count, sub_size, str_size, sizer);
    //int globalThreadId = threadIdx.x + blockIdx.x * blockDim.x;
    //printf("%d %d %d %d\n", threadIdx.x,threadIdx.y, threadIdx.z, globalThreadId );
}
void cycle_gpu(string str = "xyxxz", int n = 3) {
    if(str.length() < n) {return;}
    if(n == 0) {cout << "GPU: "<< n << endl; return;}
    int count = 0;

    int * ptr;
    hipMalloc(&ptr, sizeof(int));
    hipMemcpy(ptr, &count, sizeof(count), hipMemcpyHostToDevice);
    int size = pow(str.length() - n + 1, n);
    int sizer = factorial( str.length() ) / (factorial( n ) * factorial(  str.length() - n ));
    /*dim3 block_dim(8, 8, 8); // Задаем размер блока (x, y, z)
    dim3 grid_dim((size + block_dim.x - 1) / block_dim.x, (n + block_dim.y - 1) / block_dim.y, (n + block_dim.z - 1) / block_dim.z); // Вычисляем размер сетки (x, y, z)
    kernel<<<grid_dim, block_dim>>>( str.length(), n, ptr);*/
    //small size
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int threads_per_block = std::pow(2, str.length() - n );
    int blocks_per_grid = (size + threads_per_block - 1) / threads_per_block;

    // Ограничение на максимальное количество блоков
    // cout << blocks_per_grid << endl;
    if (blocks_per_grid > deviceProp.maxThreadsPerBlock) {
        blocks_per_grid = deviceProp.maxThreadsPerBlock;
        threads_per_block = (size + blocks_per_grid - 1) / blocks_per_grid;
    }
    dim3 block_size(
            threads_per_block,
            1,
            1
            );
    dim3 grid_size(blocks_per_grid, 1, 1);
    kernel  <<<grid_size, block_size>>> ( str.length(), n, ptr, size);
    hipDeviceSynchronize();
    hipMemcpy( &count,ptr, sizeof(int), hipMemcpyDeviceToHost);
    cout << "GPU: "<< count << endl;
    hipFree(ptr);
}


int main() {
    timespec start, finish, delta;
    clock_gettime(CLOCK_REALTIME, &start);
    cycle_gpu("xyxxzxxdaww", 3);
    clock_gettime(CLOCK_REALTIME, &finish);
    sub_timespec(start, finish, &delta);
    cout << "Execute time = " << delta.tv_sec << "," << delta.tv_nsec << " took seconds\n";
    clock_gettime(CLOCK_REALTIME, &start);
    cycle_cpu("xyxxzxxdaww", 3);
    clock_gettime(CLOCK_REALTIME, &finish);
    sub_timespec(start, finish, &delta);
    cout << "Execute time = " << delta.tv_sec << "," << delta.tv_nsec << " took seconds\n";
    //cycle_gpu("xyxxzxxdawwawda", 5);
    //cycle_cpu("xyxxzxxdawwawda", 5);
    /*
        string piece = "";
        for(int i = 0; i <= str.length() - n; i ++){
        piece = str[i];
        if (piece.length() >= n && int(piece[piece.length() - 1]) != 0){
            cout<<piece<<endl;
            count +=1;
        }
        else cycle(piece, i);
    }*/

}
