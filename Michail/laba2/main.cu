#include <iostream>
#include <string>
#include <hip/hip_runtime.h>

#include <vector>
#include <stack>

/*
        Задание 17
 Найти количество всех различных последовательностей символов заданного размера в
 заданной строке с учетом порядка следования символов (например, строка xyxxz содержит
 подстроки:
        x (3), y (1), z (1) размера 1,
        xy (1), xx (3), xz (3), yx (2), yz (1) размера 2,
        xyx (2) , xyz (1), xxx (1), xxz (3), yxx (1), yxz(2) размера 3,
        xyxx (1), xyxz (2), xxxz (1), yxxz (1) размера 4,
        xyxxz (1) размера 5;
 в скобках указано количество подстрок; других подстрок нет; таким образом, подстрок размера 1 ровно 5, размера 2 – 10, размера 3 – 10 и т.д.)
 */

using std::string;
using std::cout;
using std::cin;
using std::endl;
string str = "xyxxz";
//string str = "abcde";

int n = 3, count = 0;

int cycle_cpu(string piece, int a){
    string save = piece;
    for (int i = a + 1; i < str.length(); i++) {
        piece += str[i];
        if (piece.length() >= n ){
            cout<<piece<<endl;
            count +=1;
        }
        else cycle_cpu(piece, i);
        piece = save;
    }
}
void process_stack() {
    std::vector<int> offset (n,0);
    std::vector<int> iter (n,0);
    int maxSIZE = str.length() - n;
    string line(n,' ');
    for(int i = 0; i < n; i++) {offset[i] = i; line[i] = str[i]; }
    int c = 0;
    for(auto dd : iter ) {
        cout << dd << " ";
    }
    cout << endl;
    while(c != n) {
        if(iter.back() + 1 <= maxSIZE) {
            iter.back()++;
        }
        else {
            for (int i = 0; i < n; i++) {
                if (iter[i] >= maxSIZE) {
                    if(iter[i - 1] + 1 <= maxSIZE) {
                        iter[i - 1]++;
                    }
                    if (iter[i - 1] <= maxSIZE) {
                        iter[i] = iter[i - 1];
                    }
                    for (int j = i; j < n; j++) {
                        iter[j] = iter[i];
                    }
                }
            }
        }
        for(auto dd : iter ) {
            cout << dd << " ";
        }
        cout << endl;
        c = 0;
        for(auto d : iter) {
            if(d == maxSIZE) {
                c++;
            }
        }

    }
}
__global__ void cycle_gpu(string piece, int a) {

}

int main() {
    string piece = "";
    cycle_cpu("",-1);
    process_stack();
    /*for(int i = 0; i <= str.length() - n; i ++){
        piece = str[i];
        if (piece.length() >= n && int(piece[piece.length() - 1]) != 0){
            cout<<piece<<endl;
            count +=1;
        }
        else cycle(piece, i);
    }*/
    cout<<count;
}
